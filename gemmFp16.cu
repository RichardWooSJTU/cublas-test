#include "hip/hip_runtime.h"
#include "gemm.cu.h"
#include <fstream>
#include <iostream>

void GemmFp16(int m, int k, int n, 
    hipblasLtHandle_t ltHandle) {
    hipblasStatus_t status;
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    half * A, * B, *C;
    hipMalloc(&A, m * k * sizeof(half));
    hipMalloc(&B, k * n * sizeof(half));
    hipMalloc(&C, m * n * sizeof(half));

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};
    int lda = m, ldb = k, ldc = m;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    half alpha = (half)1.0f, beta = (half)0.0f;
    void* workspace;
    size_t workspaceSize = 4 * (m * k + m * k + n * k);
    hipMalloc(&workspace, workspaceSize);

    hipEvent_t startEvent;
    hipEvent_t stopEvent;
    if (hipEventCreate(&startEvent, hipEventBlockingSync) != hipSuccess) {
        std::cout << " hipEventCreate GG with status " << std::endl;
    }
    if (hipEventCreate(&stopEvent, hipEventBlockingSync) != hipSuccess) {
        std::cout << " hipEventCreate GG with status "<< std::endl;
    }


    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
     hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_16F, HIP_R_16F);
     hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
     hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
     hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
     hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
     hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16F, m, n, ldc);
     ////////////
    // Select //
    ////////////
    auto results = FindAlgo(ltHandle, m, n, k, A, B, C, operationDesc, Adesc, Bdesc, Cdesc, HIPBLAS_COMPUTE_16F, HIP_R_16F, HIP_R_16F, HIP_R_16F, HIP_R_16F);
    // std::ofstream outfile;
    // outfile.open("res.csv", std::ios::app);
    // int i = 0;
    // while (results[i].time == 0) i++;
    // outfile << "fp16, " << m << ", "  << k << ", " << n << ", " << results[i].time << ", " << results[i].wavesCount << ", " << results[i].workspaceSize << std::endl;

    // outfile.close();
    std::cout << "finsh select" << std::endl;

    //  hipError_t err;
    //  err = hipEventRecord(startEvent, 0);
    //  status = hipblasLtMatmul(ltHandle,
    //     operationDesc,
    //     &alpha,
    //     A,
    //     Adesc,
    //     B,
    //     Bdesc,
    //     &beta,
    //     C,
    //     Cdesc,
    //     C,
    //     Cdesc,
    //     NULL,
    //     workspace,
    //     workspaceSize,
    //     0);
    // if (status != HIPBLAS_STATUS_SUCCESS) {
    //     std::cout << "hipblasLtMatmul GG with status " << status << std::endl;
    // }
    // err = hipEventRecord(stopEvent, 0);
    // err = hipEventSynchronize(stopEvent);
    // float time;
    // err = hipEventElapsedTime(&time, startEvent, stopEvent);
    // if (err != hipSuccess) {
    //     std::cout << " cuda event elpsed time GG" << std::endl;
    // }
    // std::cout << "fp16 matmul " << time << " ms" << std::endl;;
}

void CublasGemmFp16(int m, int k, int n, hipblasHandle_t handle) {
    hipblasStatus_t status;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    const __half alpha = (__half)1.0f;
    const __half beta = (__half)0.0f;
    // float alpha = 1.0f;
    // float beta = 0.0f;
    // int32_t alpha = 1;
    // int32_t beta = 0;
    int lda = m;
    int ldb = k;
    int ldc = m;

    __half *A, *B, *C;
    hipMalloc(&A, sizeof(__half) * m * k);
    hipMalloc(&B, sizeof(__half) * n * k);
    hipMalloc(&C, sizeof(__half) * m * n);

    hipStream_t stream = 0;
    struct timeval start, end;
    gettimeofday(&start, NULL);
    const int repeats = 10000;
    for (int loop = 0; loop < repeats; loop++) {
        // Non Tensorcore
        // status = hipblasHgemm(handle,
        //     transa, transb,
        //     m, n, k,
        //     &alpha,
        //     A, lda,
        //     B, ldb,
        //     &beta,
        //     C, ldc);
        // if (status != HIPBLAS_STATUS_SUCCESS) {
        //     std::cout << "hipblasHgemm GG with status " << status << std::endl;
        //     return;
        // }

        // Use Tensorcore
        // hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;//CUBLAS_GEMM_DFALT_TENSOR_OP;
        hipblasGemmAlgo_t algo = CUBLAS_GEMM_DFALT_TENSOR_OP;

        status = hipblasGemmEx(handle,
            transa,
            transb,
            m,
            // n,
            n,
            // m,
            k,
            (void*)&alpha,
            (void*)A,
            HIP_R_16F,
            m,
            (void*)B,
            HIP_R_16F,
            k,
            (void*)&beta,
            (void*)C,
            HIP_R_16F,
            m,
            HIP_R_32F,
            algo);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "cublasHgecublasGemmExmm GG with status " << status << std::endl;
            return;
        }
    }
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    float time = diffTime(start, end);
    std::cout << "hipblasHgemm spend " << time/repeats << " ms in " << m  << ", " << k << ", " << n << std::endl;

}