#include <iostream>
#include <vector>

#include "find_algo.cu.h"

typedef struct {
    hipblasLtMatmulAlgo_t algo;
    hipblasStatus_t status;
    float time;
    size_t workspaceSize;  // actual memory workspace needed
    hipblasMath_t mathMode;
    cublasLtReductionScheme_t reductionScheme;
    int customOption;
    float wavesCount;
} customMatmulPerf_t;

const int splitKSequenceA[] = {2, 3, 4, 5, 6, 8, 12, 16, 32};

template<typename InT, typename OutT>
static hipblasStatus_t TestMatmulRun(hipblasLtHandle_t ltHandle,
                                    hipblasLtMatmulDesc_t matmulDesc,
                                    hipblasLtMatrixLayout_t A_desc,
                                    hipblasLtMatrixLayout_t B_desc,
                                    hipblasLtMatrixLayout_t C_desc,
                                    const  InT* A,
                                    const  InT* B,
                                     OutT* C,
                                    const hipblasLtMatmulAlgo_t& algo,
                                    customMatmulPerf_t& perfResults,
                                    hipEvent_t& startEvent,
                                    hipEvent_t& stopEvent
                                )
{
    hipStream_t stream = 0;
    hipblasLtMatmulHeuristicResult_t heurResult;
    hipblasStatus_t algoStatus =
        cublasLtMatmulAlgoCheck(ltHandle, matmulDesc, A_desc, B_desc, C_desc, C_desc, &algo, &heurResult);
    if (algoStatus == HIPBLAS_STATUS_SUCCESS) {
        hipError_t err;
        err = hipEventRecord(startEvent, stream);
         OutT alpha = 1, beta = 0;
        void* workSpace;
        hipMalloc(&workSpace, heurResult.workspaceSize);
        int repeats = 10;
        for (int loop = 0; loop < repeats; loop++) {
            hipblasStatus_t oneRunStatus = hipblasLtMatmul(ltHandle,
                                                            matmulDesc,
                                                            &alpha,
                                                            A,
                                                            A_desc,
                                                            B,
                                                            B_desc,
                                                            &beta,
                                                            C,
                                                            C_desc,
                                                            C,
                                                            C_desc,
                                                            &algo,
                                                            workSpace,
                                                            heurResult.workspaceSize,
                                                            stream);
            if (oneRunStatus != HIPBLAS_STATUS_SUCCESS) {
                algoStatus = oneRunStatus;
                break;
            }
        }
        err = hipEventRecord(stopEvent, stream);
        float time;
        err = hipEventElapsedTime(&time, startEvent, stopEvent);
        if (err != hipSuccess) {
            algoStatus = HIPBLAS_STATUS_INTERNAL_ERROR;
        }
        if (algoStatus == HIPBLAS_STATUS_SUCCESS) {
            perfResults.algo = algo;
            perfResults.time = time / repeats;
            perfResults.workspaceSize = heurResult.workspaceSize;
            perfResults.wavesCount = heurResult.wavesCount;
        }
    } else {
        // printf("not enough workspace! %ld\n", heurResult.workspaceSize);
        algoStatus = HIPBLAS_STATUS_NOT_SUPPORTED;  // Not enough workspace
    }
    return algoStatus;
}

template<typename InT, typename OutT>
int FindAlgo(hipblasLtHandle_t ltHandle,
             int m,
             int n,
             int k,
             const  InT* A,
             const  InT* B,
              OutT* C,
             hipblasLtMatmulDesc_t matmulDesc,
             hipblasLtMatrixLayout_t A_desc,
             hipblasLtMatrixLayout_t B_desc,
             hipblasLtMatrixLayout_t C_desc
             ) {
    // Get Ids    
    // https://docs.nvidia.com/cuda/cublas/index.html#cublasLtMatmulAlgoGetIds
    // Input
    hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32I;
    hipDataType scaleType = HIP_R_32I;
    hipDataType Atype = HIP_R_8I;
    hipDataType Btype = HIP_R_8I;
    hipDataType Ctype = HIP_R_32I;
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    // Output
    int algoIdA[100];
    int nbAlgoIds;
    status = cublasLtMatmulAlgoGetIds(
        ltHandle, computeType, scaleType, Atype, Btype, Ctype, Ctype, 100, algoIdA, &nbAlgoIds);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << " cublasLtMatmulAlgoGetIds A GG with status " << status << std::endl;
    }

    std::cout << "get " << nbAlgoIds << " algoIds" << std::endl;

    int AlgoCount = 0;
    int AlgoCombinations = 20000;
    hipblasLtMatmulAlgo_t algos[AlgoCombinations]; 
    // Loop over the Algo IDs
    for (int idx = 0; idx < nbAlgoIds; idx++) {
        std::cout << "Process algo ID " << algoIdA[idx] << std::endl;
        hipblasLtMatmulAlgo_t algo;
        
        /* Initialize algo structure with given Algp ID */
        // https://docs.nvidia.com/cuda/cublas/index.html#cublasLtMatmulAlgoInit
        status =
            cublasLtMatmulAlgoInit(ltHandle, computeType, scaleType, Atype, Btype, Ctype, Ctype, algoIdA[idx], &algo);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            std::cout << " cublasLtMatmulAlgoInit GG with status " << status << std::endl;
        }

        // Query the tiles enums supported by that algo which is used to alloc enough space to store it
        // https://docs.nvidia.com/cuda/cublas/index.html#cublasLtMatmulAlgoCapGetAttribute
        size_t sizeWritten = 0;
        cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_TILE_IDS, NULL, 0, &sizeWritten);
        int nbTiles = int(sizeWritten / sizeof(int));
        std::vector<int> tileA(nbTiles == 0 ? 1 : nbTiles);
        if (nbTiles == 0) {
            tileA[0] = CUBLASLT_MATMUL_TILE_UNDEFINED;
            nbTiles = 1;
            std::cout << "no tiles" << std::endl;
        } else {
            cublasLtMatmulAlgoCapGetAttribute(
                &algo, CUBLASLT_ALGO_CAP_TILE_IDS, tileA.data(), sizeof(int) * nbTiles, &sizeWritten);
        }
        std::cout << "has tiles " << nbTiles << std::endl;
        // Query the stages enums supported by that algo (cuda must >= 11.0)
        cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_STAGES_IDS, NULL, 0, &sizeWritten);
        int nbStages = int(sizeWritten / sizeof(int));
        std::vector<int> stagesA(nbStages == 0 ? 1 : nbStages);
        if (nbStages == 0) {
            stagesA[0] = CUBLASLT_MATMUL_STAGES_UNDEFINED;
            nbStages = 1;
            std::cout << "no stages" << std::endl;
        }
        else {
            cublasLtMatmulAlgoCapGetAttribute(
                &algo, CUBLASLT_ALGO_CAP_STAGES_IDS, stagesA.data(), sizeof(int) * nbStages, &sizeWritten);
                
        }

        std::cout << "has stages " << nbStages << std::endl;
        // Retrieve Other Algo Capabilities attributes
        int splitkSupport, redMask, swizzlingMax, customOptionMax;
        cublasLtMatmulAlgoCapGetAttribute(
            &algo, CUBLASLT_ALGO_CAP_SPLITK_SUPPORT, &splitkSupport, sizeof(splitkSupport), &sizeWritten);
        cublasLtMatmulAlgoCapGetAttribute(
            &algo, CUBLASLT_ALGO_CAP_REDUCTION_SCHEME_MASK, &redMask, sizeof(redMask), &sizeWritten);
        cublasLtMatmulAlgoCapGetAttribute(
            &algo, CUBLASLT_ALGO_CAP_CTA_SWIZZLING_SUPPORT, &swizzlingMax, sizeof(swizzlingMax), &sizeWritten);
        cublasLtMatmulAlgoCapGetAttribute(
            &algo, CUBLASLT_ALGO_CAP_CUSTOM_OPTION_MAX, &customOptionMax, sizeof(customOptionMax), &sizeWritten);

        /* Loop over the different tiles */
        for (int tileIdx = 0; tileIdx < nbTiles && AlgoCount < AlgoCombinations; tileIdx++) {
            /* Loop over different stages count */
            for (int stagesIdx = 0; stagesIdx < nbStages && AlgoCount < AlgoCombinations; stagesIdx++) {
                /* Loop over the different custom option if any */
                for (int customOption = 0; customOption <= customOptionMax && AlgoCount < AlgoCombinations; customOption++) {
                     /* Loop over the CTAs swizzling support */
                     for (int k = 0; k <= swizzlingMax && AlgoCount < AlgoCombinations; k++) {
                        int splitK_trial = 0;
                        if (splitkSupport) {
                            splitK_trial += sizeof(splitKSequenceA) / sizeof(splitKSequenceA[0]);
                        }

                        for (int l = 0; (l < (1 + splitK_trial)) && (AlgoCount < AlgoCombinations); l++) {
                            cublasLtMatmulAlgoConfigSetAttribute(
                                &algo, CUBLASLT_ALGO_CONFIG_TILE_ID, &tileA[tileIdx], sizeof(tileA[tileIdx]));
                            cublasLtMatmulAlgoConfigSetAttribute(
                                &algo, CUBLASLT_ALGO_CONFIG_STAGES_ID, &stagesA[stagesIdx], sizeof(stagesA[stagesIdx]));
                            cublasLtMatmulAlgoConfigSetAttribute(
                                &algo, CUBLASLT_ALGO_CONFIG_CUSTOM_OPTION, &customOption, sizeof(customOption));
                            cublasLtMatmulAlgoConfigSetAttribute(
                                &algo, CUBLASLT_ALGO_CONFIG_CTA_SWIZZLING, &k, sizeof(k));
                            int splitK_val = 0;
                            int redScheme = CUBLASLT_REDUCTION_SCHEME_NONE;
                            cublasLtMatmulAlgoConfigSetAttribute(
                                &algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &splitK_val, sizeof(splitK_val));
                            cublasLtMatmulAlgoConfigSetAttribute(
                                &algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &redScheme, sizeof(int));
                            if (l > 0) {  // Split-K case
                                splitK_val = splitKSequenceA[l - 1];
                                cublasLtMatmulAlgoConfigSetAttribute(&algo,
                                                                     CUBLASLT_ALGO_CONFIG_SPLITK_NUM,
                                                                     &splitKSequenceA[l - 1],
                                                                     sizeof(splitKSequenceA[l - 1]));
                                for (redScheme = 1;
                                    redScheme < (int)CUBLASLT_REDUCTION_SCHEME_MASK && (AlgoCount < AlgoCombinations);
                                    redScheme = redScheme << 1) {
                                   if (redScheme & redMask) {
                                       cublasLtMatmulAlgoConfigSetAttribute(&algo,
                                                                            CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME,
                                                                            &redScheme,
                                                                            sizeof(redScheme));

                                       hipblasLtMatmulHeuristicResult_t heurResult;
                                       hipblasStatus_t algoStatus = cublasLtMatmulAlgoCheck(
                                        ltHandle, matmulDesc, A_desc, B_desc, C_desc, C_desc, &algo, &heurResult);
                                       if (algoStatus == HIPBLAS_STATUS_SUCCESS) {
                                        std::cout << "algo " << algoIdA[idx] << " tile " << tileA[tileIdx] << " stages " << stagesA[stagesIdx]
                                            <<  " customOption "  << customOption << " k " << k  << " l " << l << " redScheme " << redScheme  << std::endl;
                                           algos[AlgoCount++] = algo;
                                       }
                                   }  // end if
                               }                    
                            } else {
                                // Prepare algos
                                hipblasLtMatmulHeuristicResult_t heurResult;
                                // https://docs.nvidia.com/cuda/cublas/index.html#cublasLtMatmulAlgoCheck
                                hipblasStatus_t algoStatus = cublasLtMatmulAlgoCheck(
                                    ltHandle, matmulDesc, A_desc, B_desc, C_desc, C_desc, &algo, &heurResult);
                                if (algoStatus == HIPBLAS_STATUS_SUCCESS) {
                                    std::cout << "algo " << algoIdA[idx] << " tile " << tileA[tileIdx] << " stages " << stagesA[stagesIdx]
                                        <<  " customOption "  << customOption << " k " << k    << std::endl;
                                    algos[AlgoCount++] = algo;
                                }
                            }
                            
                        }
                     }
                }
            }
        }
    }
    std::cout << "Got " << AlgoCount << " algos"  << std::endl;
    hipEvent_t startEvent;
    hipEvent_t stopEvent;
    std::vector<customMatmulPerf_t> perfResults(AlgoCount);
    if (hipEventCreate(&startEvent, hipEventBlockingSync) != hipSuccess) {
        std::cout << " hipEventCreate GG with status " << std::endl;
    }
    if (hipEventCreate(&stopEvent, hipEventBlockingSync) != hipSuccess) {
        std::cout << " hipEventCreate GG with status "<< std::endl;
    }
    for (int i = 0; i < AlgoCount; i++) {
        status = TestMatmulRun(ltHandle,
            matmulDesc,
            A_desc,
            B_desc,
            C_desc,
            A,
            B,
            C,
            algos[i],
            perfResults[i],
            startEvent,
            stopEvent);
        perfResults[i].status = status;
        std::cout << "algo " << i << " time " << perfResults[i].time << " wavesCount " << perfResults[i].wavesCount << " workspaceSize " << perfResults[i].workspaceSize << std::endl;

    }
}