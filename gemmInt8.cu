#include "gemm.cu.h"
#include <fstream>
#include <iostream>


void GemmInt8(int m, int k, int n, 
    hipblasLtHandle_t ltHandle
    ) {
     // Init value
     std::vector<int8_t> A_vec(m * k);
     std::vector<int8_t> B_vec(k * n);
     int8_t * A_dev, * B_dev, * A_dev_tmp, * B_dev_tmp;
     hipMalloc(reinterpret_cast<void**>(&A_dev), m * k * sizeof(int8_t));
     hipMalloc(reinterpret_cast<void**>(&B_dev), k * n * sizeof(int8_t));
     hipMalloc(reinterpret_cast<void**>(&A_dev_tmp), m * k * sizeof(int8_t));
     hipMalloc(reinterpret_cast<void**>(&B_dev_tmp), k * n * sizeof(int8_t));
 
     std::vector<int32_t> C_vec(m * n);
     int32_t * C_dev;
     hipMalloc(reinterpret_cast<void**>(&C_dev), m * n * sizeof(int32_t));
 
    //  std::cout << "origin A: "<< std::endl;
     // for (int i = 0; i < m; ++i) {
     //     for (int j = 0; j < k; ++j) {
     //         A_vec[i * k + j] = static_cast<int8_t>(j);
     //         std::cout << static_cast<int>(A_vec[i * k + j]) << " ";
     //     }
     //     std::cout << std::endl;
     // }
 
 
    //  std::cout << "origin B: "<< std::endl;
     // for (int i = 0; i < k; ++i) {
     //     for (int j = 0; j < n; ++j) {
     //         B_vec[i * n + j] = static_cast<int8_t>(j);
     //         std::cout << static_cast<int>(B_vec[i * n + j]) << " ";
     //     }
     //     std::cout << std::endl;
     // }
 
     hipMemcpy(A_dev_tmp, A_vec.data(), A_vec.size() * sizeof(A_vec[0]), hipMemcpyHostToDevice);
     hipMemcpy(B_dev_tmp, B_vec.data(), B_vec.size() * sizeof(B_vec[0]), hipMemcpyHostToDevice);
     hipblasStatus_t status;
 
     // Transpose A B
     transpose_kernelLauncher(A_dev_tmp, A_dev, m, k, 0);
     transpose_kernelLauncher(B_dev_tmp, B_dev, k, n, 0);
 
     
 
 
     // Init origin matrix desc
     hipblasLtMatrixLayout_t Adesc = NULL;
     hipblasLtMatrixLayout_t Bdesc = NULL;
     hipblasLtMatrixLayout_t Cdesc = NULL;
     hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, m);
     hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, k, n, k);
     hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, m, n, m);
    // Init matmul
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I);


    hipblasOperation_t opTranspose = HIPBLAS_OP_T;
    hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opTranspose, sizeof(opTranspose)); // opTranspose = HIPBLAS_OP_T;

    ////////////
    // Select for non IMMA//
    ////////////
    // FindAlgo(ltHandle, m, n, k, A_dev, B_dev, C_dev, matmulDesc, Adesc, Bdesc, Cdesc);
    // return 0;



    // Init transform matrix desc
    hipblasLtMatrixLayout_t ATransdesc = NULL;
    hipblasLtMatrixLayout_t BTransdesc = NULL;
    hipblasLtMatrixLayout_t CTransdesc = NULL;
    hipblasLtOrder_t order_COL32       = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t order_COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;
    hipblasLtOrder_t order_matrixB;
    order_matrixB = CUBLASLT_ORDER_COL32_2R_4R4;
    bool use_4r4 = true;
    if (use_4r4) {
        order_matrixB = CUBLASLT_ORDER_COL32_2R_4R4;
    } else {
        order_matrixB = CUBLASLT_ORDER_COL4_4R2_8C;
    }


    int ldatransform = 32 * m;
    // int ldbtransform = 32 * (k + 8 - 1) / 8 * 8; // B should be transposed
    int ldbtransform;
    if (use_4r4) {
        ldbtransform = 32 * ((n + 32 - 1) / 32) * 32;
    } else {
        ldbtransform = 32 * ((n + 8 - 1) / 8) * 8;
    }
    int ldctransform = 32 * m;
    hipblasLtMatrixLayoutCreate(&ATransdesc, HIP_R_8I, m, k, ldatransform);
    hipblasLtMatrixLayoutSetAttribute(ATransdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32));
    hipblasLtMatrixLayoutCreate(&BTransdesc, HIP_R_8I, n, k, ldbtransform);
    hipblasLtMatrixLayoutSetAttribute(BTransdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_matrixB, sizeof(order_matrixB));
    hipblasLtMatrixLayoutCreate(&CTransdesc, HIP_R_32I, m, n, ldctransform);
    hipblasLtMatrixLayoutSetAttribute(CTransdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32));

    // Transform A and B
    hipblasLtMatrixTransformDesc_t transformDesc = NULL;
    float transformAlpha = 1.0f, transformBeta = 0.0f;
    int8_t *Atransform = NULL;
    int8_t *Btransform = NULL;
    int32_t *Ctransform = NULL;
    hipMalloc(reinterpret_cast<void**>(&Atransform), sizeof(int8_t) * (k + 32 - 1) / 32 * ldatransform);
    hipMalloc(reinterpret_cast<void**>(&Btransform), sizeof(int8_t) * (k + 32 - 1) / 32 * ldbtransform);
    hipMalloc(reinterpret_cast<void**>(&Ctransform), sizeof(int32_t) * (n + 32 - 1) / 32 * ldctransform);
    hipblasLtMatrixTransformDescCreate(&transformDesc, HIP_R_32F);
    status = hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, A_dev, Adesc, &transformBeta, NULL, NULL, Atransform, ATransdesc, 0);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << " hipblasLtMatrixTransform A GG with status " << status << std::endl;
    }
    // B matrix is non-transposed, but transposed matrix is needed - add transpose operation in matrix transform.
    // HIPBLAS_OP_T 加在了哪个位置哪个位置的矩阵就是真的会转置，所以matmul的时候B设为转置位就需要先转置，再在matmul中转置回来...
    opTranspose = HIPBLAS_OP_T;
    hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)); 
    status = hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, B_dev, Bdesc, &transformBeta, NULL, NULL, Btransform, BTransdesc, 0);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "hipblasLtMatrixTransform B GG with status " << status << std::endl;
    }


    int32_t alpha = 1, beta = 0;
    // int alpha = 1, beta =  0;

    // Init algo
    hipblasLtMatmulAlgo_t algo;
    int algoId;
    algoId = 7;
    int swizzle = 0;
    int customOption = 0;
    int tile = 20;
    int splitK_val = 0;
    int reductionScheme = 0;
    status = cublasLtMatmulAlgoInit(
        ltHandle, HIPBLAS_COMPUTE_32I, HIP_R_32F, HIP_R_8I, HIP_R_8I, HIP_R_8I, HIP_R_8I, algoId, &algo);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "cublasLtMatmulAlgoInit GG with status " << status << std::endl;
    }
    cublasLtMatmulAlgoConfigSetAttribute(
        &algo, CUBLASLT_ALGO_CONFIG_CUSTOM_OPTION, &(customOption), sizeof(customOption));
    cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_TILE_ID, &(tile), sizeof(tile));
    cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &(splitK_val), sizeof(splitK_val));
    cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_CTA_SWIZZLING, &(swizzle), sizeof(swizzle));
    cublasLtMatmulAlgoConfigSetAttribute(
        &algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &(reductionScheme), sizeof(int));
    int stages;
    stages = 15;
    cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_STAGES_ID, &(stages), sizeof(stages));

    ////////////
    // Select //
    ////////////
    auto results = FindAlgo(ltHandle, m, n, k, Atransform, Btransform, Ctransform, matmulDesc, ATransdesc, BTransdesc, CTransdesc, HIPBLAS_COMPUTE_32I, HIP_R_32I, HIP_R_8I, HIP_R_8I, HIP_R_32I);
    // std::ofstream outfile;
    // outfile.open("res.csv", std::ios::app);
    // int i = 0;
    // while (results[i].time == 0) i++;
    // outfile << "int8, " << m << ", "  << k << ", " << n << ", " << results[i].time << ", " << results[i].wavesCount << ", " << results[i].workspaceSize << std::endl;

    // outfile.close();
    std::cout << "finsh select" << std::endl;
    

    status = hipblasLtMatmul(ltHandle,
        matmulDesc,
        &alpha,
        Atransform,
        ATransdesc,
        Btransform,
        BTransdesc,
        &beta,
        Ctransform,
        CTransdesc,
        Ctransform,
        CTransdesc,
        &algo,
        NULL,
        0,
        0);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "hipblasLtMatmul GG with status " << status << std::endl;
    }

    // Transform C
    opTranspose = HIPBLAS_OP_N;
    hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose));



    status = hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, Ctransform, CTransdesc, &transformBeta, NULL, NULL, C_dev, Cdesc, 0);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "hipblasLtMatrixTransform C GG with status " << status << std::endl;
    }
    checkCudaStatus(hipDeviceSynchronize());
    if (Ctransform) checkCudaStatus(hipFree(Ctransform));
    if (Btransform) checkCudaStatus(hipFree(Btransform));
    if (Atransform) checkCudaStatus(hipFree(Atransform));
    //  Watch result
    hipMemcpy(C_vec.data(), C_dev, sizeof(int32_t) * m * n, hipMemcpyDeviceToHost);
    // std::cout << "result: "<< std::endl;
    // for (int i = 0; i < m; ++i) {
    //     for (int j = 0; j < n; ++j) {
    //         std::cout << static_cast<int>(C_vec[i * n + j]) << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // hipblasHandle_t cublasH = NULL;
    // CUBLAS_CHECK(hipblasCreate(&cublasH));
    // std::cout << cublasH <<std::endl;


    // for (int i = 0; i < m * n; ++i) {
    //     std::cout << static_cast<int>(C_vec[i]) << " ";
    // }


    // A_vec.clear();
    // A_vec.resize((k + 32 - 1) / 32 * ldatransform);
    // hipMemcpy(A_vec.data(), Atransform, sizeof(int8_t) * (k + 32 - 1) / 32 * ldatransform, hipMemcpyDeviceToHost);

    // B_vec.clear();
    // B_vec.resize((k + 32 - 1) / 32 * ldbtransform);
    // hipMemcpy(B_vec.data(), Btransform, sizeof(int8_t) * (k + 32 - 1) / 32 * ldbtransform, hipMemcpyDeviceToHost);


    // std::cout << "after transform A: "<< std::endl;
    // // for (int i = 0; i < m; ++i) {
    // //     for (int j = 0; j < k; ++j) {
    // //         std::cout << static_cast<int>(A_host[i * k + j]) << " ";
    // //     }
    // //     std::cout << std::endl;
    // // }
    // for (int i = 0; i < (k + 32 - 1) / 32 * ldatransform; ++i) std::cout << static_cast<int>(A_vec[i]) << " ";

    // std::cout << std::endl;

    // std::cout << "after transform B: "<< std::endl;
    // for (int i = 0; i < (k + 32 - 1) / 32 * ldbtransform; ++i) std::cout << static_cast<int>(B_vec[i]) << " ";

}


void CublasGemmInt8(int m, int k, int n, hipblasHandle_t handle) {
    hipblasStatus_t status;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_T;
    const int32_t alpha = (int32_t)1;
    const int32_t beta = (int32_t)0;
    // float alpha = 1.0f;
    // float beta = 0.0f;
    // int32_t alpha = 1;
    // int32_t beta = 0;
    int lda = 32 * m;
    int ldb =  32 * ((n + 32 - 1) / 32) * 32;
    int ldc = 32 * m;



    int32_t *A, *B, *C;
    hipMalloc(&A, sizeof(int32_t) * m * k);
    hipMalloc(&B, sizeof(int32_t) * n * k);
    hipMalloc(&C, sizeof(int32_t) * m * n);

    hipStream_t stream = 0;
    struct timeval start, end;
    gettimeofday(&start, NULL);
    const int repeats = 10000;
    for (int loop = 0; loop < repeats; loop++) {

        // Use Tensorcore
        // hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;//CUBLAS_GEMM_DFALT_TENSOR_OP;
        hipblasGemmAlgo_t algo = CUBLAS_GEMM_DFALT_TENSOR_OP;

        status = hipblasGemmEx(handle,
            transa,
            transb,
            m,
            // n,
            n,
            // m,
            k,
            (void*)&alpha,
            (void*)A,
            HIP_R_8I,
            lda,
            (void*)B,
            HIP_R_8I,
            ldb,
            (void*)&beta,
            (void*)C,
            HIP_R_32I,
            ldc,
            HIPBLAS_COMPUTE_32I,
            algo);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "hipblasGemmEx GG with status " << status << std::endl;
            return;
        }
    }
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    float time = diffTime(start, end);
    std::cout << "hipblasHgemm spend " << time/repeats << " ms in " << m  << ", " << k << ", " << n << std::endl;

}