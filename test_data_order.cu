#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <fstream>

#include "cublas.cu.h"
#include "gemm.cu.h"
#include "gemm.h"

__global__ void transpose(int8_t * src, int8_t* dst, int m, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = tid / n;
    int j = tid % n;
    
    dst[j*m+i] = src[i * n + j];
    
}

template<typename T>
std::vector<T> OpenInput(int m, int n, std::string name) {
    std::vector<T> res(m * n);
    std::ifstream infile;
    infile.open(name+".txt", std::ios::in);
    for (int i = 0; i < m * n; ++i) {
        int tmp;
        infile >> tmp;
        // std::cout << static_cast<int>(tmp) << std::endl;
        res[i] = tmp;
    }
    infile.close();
    return res;
}


int main() {
    //  std::vector<std::vector<int>> params{{4096, 12288}, {4096, 16384}, {4096, 4096}, {16384, 4096}, {256, 768}, {256, 1024}, {1024, 256}, {256, 256}};
    hipblasLtHandle_t ltHandle;
    hipblasLtCreate(&ltHandle);
    hipblasHandle_t handle;
     hipblasCreate(&handle);
    //  std::vector<std::vector<int>> params{{32, 32}};
      std::vector<std::vector<int>> params{{768, 768}};
    //  int m = 4;
    // for (int m : {1}) {
    //      for (auto p : params) {
            // Init cublasLt
            int m = 1, k = 768, n = 768;
            // int k = p[0], n = p[1];
            auto A = OpenInput<int8_t>(m, k, "input");
            auto B = OpenInput<int8_t>(k, n, "weight");
            auto C_1 = GemmInt8Imma1(A, B, m, n, k);

            std::vector<int32_t> C_2(m*n);
            Gemm(A.data(), B.data(), C_2.data(), m, n, k);



            
            // auto C_v1 =  GemmInt8(m, k, n, ltHandle);
            //auto C_v1 =  GemmInt8(n, k, m, ltHandle);
            // GemmFp16(m, k, n, ltHandle);
            // CublasGemmFp16(m, k, n, handle);
            // std::vector<int8_t> A_v(m*k), B_v(k*n);
            // std::vector<int32_t> C_v1(m*n), C_v2(m*n);
            // auto C_v2 = CublasGemmInt8(m, k, n, handle);
            //CublasGemmInt8(n, k, m, handle, B_v.data(), A_v.data(), C_v1.data());
            // Gemm(handle, A_v.data(), B_v.data(), C_v2.data(), m, n, k);
            bool f = true;
            for (int i = 0; i < m; ++i) {
                for (int j = 0; j < n; ++j) {
                    if (C_1[i*n+j] != C_2[i*n+j]) {
                        std::cout << i << " " << j  << " "<< C_1[i*n+j] << " != " << C_2[i*n+j] << std::endl;
                        f = false;
                    }
                }
            }
            if (f) {
                std::cout << "Congratulations!!!" << std::endl;
            }
            
    //     }
    // }
    
//CublasGemmInt8(2, 4, 2, handle);
    


}